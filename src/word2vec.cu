#include "hip/hip_runtime.h"
#include <cstdio>
#include <queue>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include "type.hpp"
#include <vector>
#include <stdexcept>
#include <string>
#include <iostream>
#include <thread>
#include <chrono>
#include <unistd.h>
#include "edge_container.hpp"
#include "lr_scheduler.hpp"
#include <algorithm>

using std::vector;
using std::string;
using std::cout;
using std::endl;

#define DELTA_R  100
#define MAX_STRING 100
#define EXP_TABLE_SIZE 1000
#define MAX_EXP 6
#define MAX_SENTENCE_LENGTH 1000
#define MAX_CODE_LENGTH 40

#define MAX_SENTENCE 15000
#define checkCUDAerr(err) {\
  hipError_t cet = err;\
  if (hipSuccess != cet) {\
    printf("%s %d : %s\n", __FILE__, __LINE__, hipGetErrorString(cet));\
    exit(0);\
  }\
}


const int vocab_hash_size = 30000000;  // Maximum 30 * 0.7 = 21M words in the vocabulary

struct vocab_word {
  long long cn;
  int *point;
  char *word, *code, codelen;
};

int my_rank;
float *last_emb;

char train_file[MAX_STRING], output_file[MAX_STRING];
char save_vocab_file[MAX_STRING], read_vocab_file[MAX_STRING];
struct vocab_word *vocab;
int binary = 0, cbow = 1, debug_mode = 2, window = 5, min_count = 5, min_reduce = 1, reuseNeg = 1;
int *vocab_hash;
long long vocab_max_size = 1000, vocab_size = 0, layer1_size = 100;
long long train_words = 0, word_count_actual = 0, iter = 5, file_size = 0, classes = 0;
float alpha = 0.025, starting_alpha, sample = 1e-3;
float *syn0, *syn1, *syn1neg, *expTable;
clock_t start;

int hs = 0, negative = 5;
const int table_size = 1e8;
int *table;

// FOR CUDA
int *vocab_codelen, *vocab_point, *d_vocab_codelen, *d_vocab_point;
char *vocab_code, *d_vocab_code;
int *d_table;
float *d_syn0, *d_syn1, *d_expTable;

__device__ float reduceInWarp(float f) {
  for (int i=warpSize/2; i>0; i/=2) {
    f += __shfl_sync(0xFFFFFFFF, f, i, 32);
  }
  return f;
}

__device__ void warpReduce(volatile float* sdata, int tid) {
  sdata[tid] += sdata[tid + 32];
  sdata[tid] += sdata[tid + 16];
  sdata[tid] += sdata[tid + 8];
  sdata[tid] += sdata[tid + 4];
  sdata[tid] += sdata[tid + 2];
  sdata[tid] += sdata[tid + 1];
}

// calculate cosine similarity of all nodes themselves
__global__ void cosine_similarity_kernel(float *d_vectors, float *d_result, long long  v, int dim){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
  // printf("thread [%d,%d]\n",i,j);
	if( i < v && j < v) {
		float dot_product = 0.0f;
		float norm_i = 0.0f;
		float norm_j = 0.0f;

		// calculate dot_product and L2 norm.
		for (int k = 0; k < dim; ++k) {
			float vec_i = d_vectors[i * dim + k];
      float vec_j = d_vectors[j * dim + k];
			dot_product += vec_i * vec_j;
			norm_i += vec_i * vec_i;
			norm_j += vec_j * vec_j;	
		}
		norm_i = sqrt(norm_i);
		norm_j = sqrt(norm_j);
		// calculate similarity
		if(norm_i > 0.0f && norm_j > 0.0f) {
			d_result[i * v + j] = dot_product / (norm_i * norm_j);
		}else {
			d_result[i * v + j] = 0.0f; // prevent division by zero
		}
	}

}

__device__ float sigmoid(float x) {
	return 1.0f / (1.0f + expf(-x));
}


__global__ void compute_kl_divergence_kernel(float *d_A,float *d_B, float *d_result, long long  v){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if( idx < v * v) {
		float p = sigmoid(d_A[idx]);
		float q = sigmoid(d_B[idx]);
    // printf("sigmid(%d): %f, %f\n",idx,p,q);
		// 计算相对熵的部分贡献
		if(p > 0.0f && q > 0.0f){
			float contribution = p * logf(p /q );
      // printf("compute kl idx: %d val: %.2f\n",idx, contribution);
			atomicAdd(d_result, contribution); //  sum up
		}
	}
}

void  compute_kl_node_and_emb(float *h_node_sim, float *h_emb,float* h_result, long long  v,int dim){
	float *d_node_sim, *d_emb, *d_cosine, *d_result;
	hipMalloc((void**)&d_emb, v * dim * sizeof(float));	
	hipMalloc((void**)&d_node_sim, v * v * sizeof(float));	
	hipMalloc((void**)&d_cosine, v * v * sizeof(float));	
	hipMalloc((void**)&d_result, sizeof(float));
	
	hipMemset(d_result, 0, sizeof(float));

	hipMemcpy(d_node_sim,h_node_sim, v * v * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_emb,h_emb, v * dim * sizeof(float), hipMemcpyHostToDevice);

	dim3 blockSize(256);
	dim3 gridSize((v*v + blockSize.x - 1) / blockSize.x);

	// d_cosine 
	cosine_similarity_kernel<<<gridSize, blockSize>>>(d_emb,d_cosine,v,dim);

	// kl_divergence
	compute_kl_divergence_kernel<<<gridSize,blockSize>>>(d_node_sim,d_cosine,d_result,v);

	// copy result from device to host
	hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_node_sim);
	hipFree(d_emb);
	hipFree(d_cosine);
	hipFree(d_result);
}
void write2file(float* ptr,size_t size,char* filename){
	FILE* f = fopen(filename,"w");
	if(f==NULL){
		printf("Failed to open %s\n",filename);
		return;
	}
	for(size_t i = 0;i<size; i++){
		fprintf(f,"%.3f ",ptr[i]);
	}
	fclose(f);
}

void  compute_kl_from_emb(float *emb1, float *emb2,float* h_result, long long v,int dim){
	float *d_emb1, *d_emb2, *d_cosine1, *d_cosine2, *d_result;
	hipMalloc((void**)&d_emb1, v * dim * sizeof(float));	
	hipMalloc((void**)&d_emb2, v * dim * sizeof(float));	
	hipMalloc((void**)&d_cosine1, (size_t)v * v * sizeof(float));	
	hipMalloc((void**)&d_cosine2, (size_t)v * v * sizeof(float));	
	hipMalloc((void**)&d_result, sizeof(float));
	
	hipMemset(d_result, 0, sizeof(float));


	hipMemcpy(d_emb1,emb1, v * dim * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_emb2,emb2, v * dim * sizeof(float), hipMemcpyHostToDevice);

	dim3 blockSize(16,16);
	dim3 gridSize((v+blockSize.x-1) / blockSize.x,(v+blockSize.y-1)/blockSize.y);

	// d_cosine 
	cosine_similarity_kernel<<<gridSize, blockSize>>>(d_emb1,d_cosine1,v,dim);
	cosine_similarity_kernel<<<gridSize, blockSize>>>(d_emb2,d_cosine2,v,dim);

	float *h_cosine = (float*)malloc( (size_t)v * v * sizeof(float));
	if(h_cosine == NULL)printf("Failed to allocate Mem\n");
  hipDeviceSynchronize();
	hipMemcpy(h_cosine,d_cosine1, (size_t)v * v * sizeof(float), hipMemcpyDeviceToHost);
	write2file(h_cosine,(size_t)v * v,"cosine1.txt");
	hipMemcpy(h_cosine,d_cosine2, (size_t)v * v  * sizeof(float), hipMemcpyDeviceToHost);
	write2file(h_cosine,(size_t)v* v,"cosine2.txt");

	int kl_blockSize = 256;
  int kl_gridSize = (v * v + kl_blockSize - 1) / kl_blockSize;
	// kl_divergence
	compute_kl_divergence_kernel<<<kl_gridSize,kl_blockSize>>>(d_cosine1,d_cosine2,d_result,v);
  hipDeviceSynchronize();

	// copy result from device to host
	hipMemcpy(h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

	hipFree(d_emb1);
	hipFree(d_emb2);
	hipFree(d_cosine1);
	hipFree(d_cosine2);
	hipFree(d_result);
}


template<unsigned int VSIZE>
__global__ void __sgNegReuse(const int window, const int layer1_size, const int negative, const int vocab_size, float alpha,
    const int* __restrict__ sen, const int* __restrict__ sentence_length,
    float *syn1, float *syn0, const int *negSample)
{
  __shared__ float neu1e[VSIZE];

  const int sentIdx_s = sentence_length[blockIdx.x];
  const int sentIdx_e = sentence_length[blockIdx.x + 1];
  const int tid = threadIdx.x + blockDim.x * threadIdx.y;
  const int dxy = blockDim.x * blockDim.y;

  int _negSample;
  if (threadIdx.y < negative) {                                         // Get the negative sample
    _negSample = negSample[blockIdx.x * negative + threadIdx.y];
  }

  for (int sentPos = sentIdx_s; sentPos < sentIdx_e; sentPos++) {
    int word = sen[sentPos];                                            // Target word
    if (word == -1) continue;

    for (int a=0; a<window*2+1; a++) if (a != window) {
      int c = sentPos - window + a;                                     // The index of context word
      if (c >= sentIdx_s && c < sentIdx_e && sen[c] != -1) {
        int l1 = sen[c] * layer1_size;

        for (int i=tid; i<layer1_size; i+=dxy) {
          neu1e[i] = 0;
        }
        __syncthreads();

        int target, label, l2;
        float f = 0, g;
        if (threadIdx.y == negative) {                                  // Positive sample
          target = word;
          label = 1;
        } else {                                                        // Negative samples
          if (_negSample == word) goto NEGOUT;
          target = _negSample;
          label = 0;
        }
        l2 = target * layer1_size;

        for (int i=threadIdx.x; i<layer1_size; i+=blockDim.x) {         // Get gradient
          f += syn0[i + l1] * syn1[i + l2];
        }
        f = reduceInWarp(f);
        if      (f >  MAX_EXP) g = (label - 1) * alpha;
        else if (f < -MAX_EXP) g = (label - 0) * alpha;
        else {
          int tInt = (int)((f + MAX_EXP) * (EXP_TABLE_SIZE / MAX_EXP / 2));
          float t = exp((tInt / (float)EXP_TABLE_SIZE * 2 - 1) * MAX_EXP);
          t = t / (t + 1);
          g = (label - t) * alpha;
        }
        
        for (int i=threadIdx.x; i<layer1_size; i+=warpSize) {
          atomicAdd(&neu1e[i], g * syn1[i + l2]);
        }
        for (int i=threadIdx.x; i<layer1_size; i+=warpSize) {           // Update syn1 of negative sample
          syn1[i + l2] += g * syn0[i + l1];
        }

NEGOUT:
        __syncthreads();

        for (int i=tid; i<layer1_size; i+=dxy) {                        // Update syn0 of context word
          atomicAdd(&syn0[i + l1], neu1e[i]);
        }
      }
    }
  }
}

template<unsigned int FSIZE>
__global__ void skip_gram_kernel(int window, int layer1_size, int negative, int hs, int table_size, int vocab_size, float alpha,
    const float* __restrict__ expTable, const int* __restrict__ table, 
    const int* __restrict__ vocab_codelen, const int* __restrict__ vocab_point, const char* __restrict__ vocab_code,
    const int* __restrict__ sen, const int* __restrict__ sentence_length, float *syn1, float *syn0)
{
  __shared__ float f[FSIZE], g;

  int sent_idx_s = sentence_length[blockIdx.x];
  int sent_idx_e = sentence_length[blockIdx.x + 1]; 
  unsigned long next_random = blockIdx.x;

  if (threadIdx.x < layer1_size) for (int sentence_position = sent_idx_s; sentence_position < sent_idx_e; sentence_position++) {
    int word = sen[sentence_position];
    if (word == -1) continue;
    float neu1e = 0;
    next_random = next_random * (unsigned long)2514903917 + 11; 
    int b = next_random % window;

    for (int a = b; a < window * 2 + 1 - b; a++) if (a != window) {
      int c = sentence_position - window + a;
      if (c <  sent_idx_s) continue;
      if (c >= sent_idx_e) continue;
      int last_word = sen[c];
      if (last_word == -1) continue;
      int l1 = last_word * layer1_size;
      neu1e = 0;

      // HIERARCHICAL SOFTMAX
      if (hs) for (int d = vocab_codelen[word]; d < vocab_codelen[word+1]; d++) {
        int l2 = vocab_point[d] * layer1_size;

        if (threadIdx.x <  FSIZE) f[threadIdx.x] = syn0[threadIdx.x + l1] * syn1[threadIdx.x + l2];
        __syncthreads();
        if (threadIdx.x >= FSIZE) f[threadIdx.x%(FSIZE)] += syn0[threadIdx.x + l1] * syn1[threadIdx.x + l2];
        __syncthreads();
        for (int i=(FSIZE/2); i>0; i/=2) {
          if (threadIdx.x < i) f[threadIdx.x] += f[i + threadIdx.x];
          __syncthreads();
        }

        if      (f[0] <= -MAX_EXP) continue;
        else if (f[0] >=  MAX_EXP) continue;
        else if (threadIdx.x == 0) {
          f[0] = expTable[(int)((f[0] + MAX_EXP) * (EXP_TABLE_SIZE / MAX_EXP / 2))];
          g = (1 - vocab_code[d] - f[0]) * alpha;
        }
        __syncthreads();

        neu1e += g * syn1[threadIdx.x + l2];
        atomicAdd(&syn1[threadIdx.x + l2], g * syn0[threadIdx.x + l1]);
      }

      // NEGATIVE SAMPLING
      if (negative > 0) for (int d = 0; d < negative + 1; d++) {
        int target, label;
        if (d == 0) {
          target = word;
          label = 1;
        } else {
          next_random = next_random * (unsigned long)25214903917 + 11; 
          target = table[(next_random >> 16) % table_size];
          if (target == 0)    target = next_random % (vocab_size - 1) + 1;
          if (target == word) continue;
          label = 0;
        }
        int l2 = target * layer1_size;

        if (threadIdx.x <  FSIZE) f[threadIdx.x] = syn0[threadIdx.x +l1] * syn1[threadIdx.x + l2];
        __syncthreads();
        if (threadIdx.x >= FSIZE) f[threadIdx.x%(FSIZE)] += syn0[threadIdx.x + l1] * syn1[threadIdx.x + l2];
        __syncthreads();
        for (int i=(FSIZE/2); i>0; i/=2) {
          if (threadIdx.x < i)
            f[threadIdx.x] += f[i + threadIdx.x];
          __syncthreads();
        }
        if (threadIdx.x == 0) {
          if (f[0] >  MAX_EXP)
            g = (label - 1) * alpha;
          else if (f[0] < -MAX_EXP)
            g = (label - 0) * alpha;
          else
            g = (label - expTable[(int)((f[0]+MAX_EXP) * (EXP_TABLE_SIZE / MAX_EXP / 2))]) * alpha;
        }
        __syncthreads();

        neu1e += g * syn1[threadIdx.x + l2];
        atomicAdd(&syn1[threadIdx.x + l2], g * syn0[threadIdx.x + l1]);
      }

      atomicAdd(&syn0[threadIdx.x + l1], neu1e);
    }
  }
}

template<unsigned int FSIZE>
__global__ void cbow_kernel(int window, int layer1_size, int negative, int hs, int table_size, int vocab_size, float alpha,
    const float* __restrict__ expTable, const int* __restrict__ table,
    const int* __restrict__ vocab_codelen, const int* __restrict__ vocab_point, const char* __restrict__ vocab_code,
    const int* __restrict__ sen, const int* __restrict__ sentence_length, float *syn1, float *syn0)
{
  __shared__ float f[FSIZE], g;

  int sent_idx_s = sentence_length[blockIdx.x];
  int sent_idx_e = sentence_length[blockIdx.x + 1];
  unsigned long next_random = blockIdx.x;

  if (threadIdx.x < layer1_size) for (int sentence_position = sent_idx_s; sentence_position < sent_idx_e; sentence_position++) {
    int word = sen[sentence_position];
    if (word == -1) continue;
    float neu1 = 0;
    float neu1e = 0;
    next_random = next_random * (unsigned long)2514903917 + 11;
    int b = next_random % window;

    int cw = 0;
    for (int a = b; a < window * 2 + 1 - b; a++) if (a != window) {
      int c = sentence_position - window + a;
      if (c <  sent_idx_s) continue;
      if (c >= sent_idx_e) continue;
      int last_word = sen[c];
      if (last_word == -1) continue;
      neu1 += syn0[last_word * layer1_size + threadIdx.x];
      cw++;
    }

    if (cw) {
      neu1 /= cw;

      // HIERARCHICAL SOFTMAX
      if (hs) for (int d = vocab_codelen[word]; d < vocab_codelen[word+1]; d++) {
        int l2 = vocab_point[d] * layer1_size;

        if (threadIdx.x <  FSIZE) f[threadIdx.x] = neu1 * syn1[threadIdx.x + l2];
        __syncthreads();
        if (threadIdx.x >= FSIZE) f[threadIdx.x%(FSIZE)] += neu1 * syn1[threadIdx.x + l2];
        __syncthreads();
        for (int i=(FSIZE/2); i>0; i/=2) {
          if (threadIdx.x < i)
            f[threadIdx.x] += f[i + threadIdx.x];
          __syncthreads();
        }

        if      (f[0] <= -MAX_EXP) continue;
        else if (f[0] >=  MAX_EXP) continue;
        else if (threadIdx.x == 0) {
          f[0] = expTable[(int)((f[0] + MAX_EXP) * (EXP_TABLE_SIZE / MAX_EXP / 2))];
          g = (1 - vocab_code[d] - f[0]) * alpha;
        }
        __syncthreads();

        neu1e += g * syn1[threadIdx.x + l2];
        atomicAdd(&syn1[threadIdx.x + l2], g * neu1);
      }

      // NEGATIVE SAMPLING
      if (negative > 0) for (int d = 0; d < negative + 1; d++) {
        int target, label;
        if (d == 0) {
          target = word;
          label = 1;
        } else {
          next_random = next_random * (unsigned long)25214903917 + 11;
          target = table[(next_random >> 16) % table_size];
          if (target==0)    target = next_random % (vocab_size - 1) + 1;
          if (target==word) continue;
          label = 0;
        }
        int l2 = target * layer1_size;

        if (threadIdx.x <  FSIZE) f[threadIdx.x] = neu1 * syn1[threadIdx.x + l2];
        __syncthreads();
        if (threadIdx.x >= FSIZE) f[threadIdx.x%(FSIZE)] += neu1 * syn1[threadIdx.x + l2];
        __syncthreads();
        for (int i=(FSIZE/2); i>0; i/=2) {
          if (threadIdx.x < i)
            f[threadIdx.x] += f[i + threadIdx.x];
          __syncthreads();
        }
        if (threadIdx.x == 0) {
          if (f[0] > MAX_EXP)
            g = (label - 1) * alpha;
          else if (f[0] < -MAX_EXP)
            g = (label - 0) * alpha;
          else
            g = (label - expTable[(int)((f[0]+MAX_EXP) * (EXP_TABLE_SIZE / MAX_EXP / 2))]) * alpha;
        }
        __syncthreads();

        neu1e += g * syn1[l2 + threadIdx.x];
        atomicAdd(&syn1[l2 + threadIdx.x], g * neu1);
      }

      for (int a = b; a < window * 2 + 1 - b; a++) if (a != window) {
        int c = sentence_position - window + a;
        if (c <  sent_idx_s) continue;
        if (c >= sent_idx_e) continue;
        int last_word = sen[c];
        if (last_word == -1) continue;
        atomicAdd(&syn0[last_word * layer1_size + threadIdx.x], neu1e);
      }
    }
  }
}

void InitVocabStructCUDA()
{
  vocab_codelen = (int *)malloc((vocab_size + 1) * sizeof(int));
  vocab_codelen[0] = 0;
  for (int i = 1; i < vocab_size + 1; i++) 
    vocab_codelen[i] = vocab_codelen[i-1] + vocab[i-1].codelen;
  vocab_point = (int *)malloc(vocab_codelen[vocab_size] * sizeof(int));
  vocab_code = (char *)malloc(vocab_codelen[vocab_size] * sizeof(char));

  checkCUDAerr(hipMalloc((void **)&d_vocab_codelen, (vocab_size + 1) * sizeof(int)));
  checkCUDAerr(hipMalloc((void **)&d_vocab_point, vocab_codelen[vocab_size] * sizeof(int)));
  checkCUDAerr(hipMalloc((void **)&d_vocab_code, vocab_codelen[vocab_size] * sizeof(char)));

  for (int i=0; i<vocab_size; i++) {
    for (int j=0; j<vocab[i].codelen; j++) {
      vocab_code[vocab_codelen[i] + j] = vocab[i].code[j];
      vocab_point[vocab_codelen[i] + j] = vocab[i].point[j];
    }   
  }   

  checkCUDAerr(hipMemcpy(d_vocab_codelen, vocab_codelen, (vocab_size + 1) * sizeof(int), hipMemcpyHostToDevice));
  checkCUDAerr(hipMemcpy(d_vocab_point, vocab_point, vocab_codelen[vocab_size] * sizeof(int), hipMemcpyHostToDevice));
  checkCUDAerr(hipMemcpy(d_vocab_code, vocab_code, vocab_codelen[vocab_size] * sizeof(char), hipMemcpyHostToDevice));
}


void InitUnigramTable() {
  int a, i;
  double train_words_pow = 0;
  double d1, power = 0.75;
  table = (int *)malloc(table_size * sizeof(int));
  for (a = 0; a < vocab_size; a++) train_words_pow += pow(vocab[a].cn, power);
  i = 0;
  d1 = pow(vocab[i].cn, power) / train_words_pow;
  for (a = 0; a < table_size; a++) {
    table[a] = i;
    if (a / (double)table_size > d1) {
      i++;
      d1 += pow(vocab[i].cn, power) / train_words_pow;
    }
    if (i >= vocab_size) i = vocab_size - 1;
  }
  // FOR CUDA
  checkCUDAerr(hipMalloc((void **)&d_table, table_size*sizeof(int)));
  checkCUDAerr(hipMemcpy(d_table, table, table_size*sizeof(int), hipMemcpyHostToDevice));
}

// Reads a single word from a file, assuming space + tab + EOL to be word boundaries
void ReadWord(char *word, FILE *fin) {
  int a = 0, ch;
  while (!feof(fin)) {
    ch = fgetc(fin);
    if (ch == 13) continue;
    if ((ch == ' ') || (ch == '\t') || (ch == '\n')) {
      if (a > 0) { if (ch == '\n') ungetc(ch, fin);
        break;
      }
      if (ch == '\n') {
        strcpy(word, (char *)"</s>");
        return;
      } else continue;
    }
    word[a] = ch;
    a++;
    if (a >= MAX_STRING - 1) a--;   // Truncate too long words
  }
  word[a] = 0;
}

// Returns hash value of a word
int GetWordHash(char *word) {
  unsigned long long a, hash = 0;
  for (a = 0; a < strlen(word); a++) hash = hash * 257 + word[a];
  hash = hash % vocab_hash_size;
  return hash;
}

// Returns position of a word in the vocabulary; if the word is not found, returns -1
int SearchVocab(char *word) {
  unsigned int hash = GetWordHash(word);
  while (1) {
    if (vocab_hash[hash] == -1) return -1;
    if (!strcmp(word, vocab[vocab_hash[hash]].word)) return vocab_hash[hash];
    hash = (hash + 1) % vocab_hash_size;
  }
  //  return -1;
}

// Reads a word and returns its index in the vocabulary
int ReadWordIndex(FILE *fin) {
  char word[MAX_STRING];
  ReadWord(word, fin);
  if (feof(fin)) return -1;
  return SearchVocab(word);
}

// Adds a word to the vocabulary
int AddWordToVocab(char *word) {
  unsigned int hash, length = strlen(word) + 1;
  if (length > MAX_STRING) length = MAX_STRING;
  vocab[vocab_size].word = (char *)calloc(length, sizeof(char));
  strcpy(vocab[vocab_size].word, word);
  vocab[vocab_size].cn = 0;
  vocab_size++;
  // reallocate memory if needed
  if (vocab_size + 2 >= vocab_max_size) {
    vocab_max_size += 1000;
    vocab = (struct vocab_word *)realloc(vocab, vocab_max_size * sizeof(struct vocab_word));
  }
  hash = GetWordHash(word);
  while (vocab_hash[hash] != -1) hash = (hash + 1) % vocab_hash_size;
  vocab_hash[hash] = vocab_size - 1;
  return vocab_size - 1;
}

// Used later for sorting by word counts
int VocabCompare(const void *a, const void *b) {
  return ((struct vocab_word *)b)->cn - ((struct vocab_word *)a)->cn;
}

// Sorts the vocabulary by frequency using word counts
void SortVocab() {
  int a, size;
  unsigned int hash;
  // Sort the vocabulary and keep </s> at the first position
  qsort(&vocab[0], vocab_size, sizeof(struct vocab_word), VocabCompare);
  for (a = 0; a < vocab_hash_size; a++) vocab_hash[a] = -1;
  size = vocab_size;
  train_words = 0;
  for (a = 0; a < size; a++) {
    // Words occuring less than min_count times will be discarded from the vocab
    if ((vocab[a].cn < min_count) && (a != 0)) {
      vocab_size--;
      free(vocab[a].word);
    } else {
      // Hash will be re-computed, as after the sorting it is not actual
      hash=GetWordHash(vocab[a].word);
      while (vocab_hash[hash] != -1) hash = (hash + 1) % vocab_hash_size;
      vocab_hash[hash] = a;
      train_words += vocab[a].cn;
    }
  }
  vocab = (struct vocab_word *)realloc(vocab, (vocab_size + 1) * sizeof(struct vocab_word));
  // Allocate memory for the binary tree construction
  for (a = 0; a < vocab_size; a++) {
    vocab[a].code = (char *)calloc(MAX_CODE_LENGTH, sizeof(char));
    vocab[a].point = (int *)calloc(MAX_CODE_LENGTH, sizeof(int));
  }
}

// Reduces the vocabulary by removing infrequent tokens
void ReduceVocab() {
  int a, b = 0;
  unsigned int hash;
  for (a = 0; a < vocab_size; a++) if (vocab[a].cn > min_reduce) {
    vocab[b].cn = vocab[a].cn;
    vocab[b].word = vocab[a].word;
    b++;
  } else free(vocab[a].word);
  vocab_size = b;
  for (a = 0; a < vocab_hash_size; a++) vocab_hash[a] = -1;
  for (a = 0; a < vocab_size; a++) {
    // Hash will be re-computed, as it is not actual
    hash = GetWordHash(vocab[a].word);
    while (vocab_hash[hash] != -1) hash = (hash + 1) % vocab_hash_size;
    vocab_hash[hash] = a;
  }
  fflush(stdout);
  min_reduce++;
}

// Create binary Huffman tree using the word counts
// Frequent words will have short uniqe binary codes
void CreateBinaryTree() {
  long long a, b, i, min1i, min2i, pos1, pos2, point[MAX_CODE_LENGTH];
  char code[MAX_CODE_LENGTH];
  long long *count = (long long *)calloc(vocab_size * 2 + 1, sizeof(long long));
  long long *binary = (long long *)calloc(vocab_size * 2 + 1, sizeof(long long));
  long long *parent_node = (long long *)calloc(vocab_size * 2 + 1, sizeof(long long));
  for (a = 0; a < vocab_size; a++) count[a] = vocab[a].cn;
  for (a = vocab_size; a < vocab_size * 2; a++) count[a] = 1e15;
  pos1 = vocab_size - 1;
  pos2 = vocab_size;
  // Following algorithm constructs the Huffman tree by adding one node at a time
  for (a = 0; a < vocab_size - 1; a++) {
    // First, find two smallest nodes 'min1, min2'
    if (pos1 >= 0) {
      if (count[pos1] < count[pos2]) {
        min1i = pos1;
        pos1--;
      } else {
        min1i = pos2;
        pos2++;
      }
    } else {
      min1i = pos2;
      pos2++;
    }
    if (pos1 >= 0) {
      if (count[pos1] < count[pos2]) {
        min2i = pos1;
        pos1--;
      } else {
        min2i = pos2;
        pos2++;
      }
    } else {
      min2i = pos2;
      pos2++;
    }
    count[vocab_size + a] = count[min1i] + count[min2i];
    parent_node[min1i] = vocab_size + a;
    parent_node[min2i] = vocab_size + a;
    binary[min2i] = 1;
  }
  // Now assign binary code to each vocabulary word
  for (a = 0; a < vocab_size; a++) {
    b = a;
    i = 0;
    while (1) {
      code[i] = binary[b];
      point[i] = b;
      i++;
      b = parent_node[b];
      if (b == vocab_size * 2 - 2) break;
    }
    vocab[a].codelen = i;
    vocab[a].point[0] = vocab_size - 2;
    for (b = 0; b < i; b++) {
      vocab[a].code[i - b - 1] = code[b];
      vocab[a].point[i - b] = point[b] - vocab_size;
    }
  }
  free(count);
  free(binary);
  free(parent_node);
}

void LearnVocabFromTrainFile() {
  char word[MAX_STRING];
  FILE *fin;
  long long a, i;
  for (a = 0; a < vocab_hash_size; a++) vocab_hash[a] = -1;
  fin = fopen(train_file, "rb");
  if (fin == NULL) {
    printf("ERROR: training data file not found!\n");
    exit(1);
  }
  vocab_size = 0;
  AddWordToVocab((char *)"</s>");
  while (1) {
    ReadWord(word, fin);
    if (feof(fin)) break;
    train_words++;
    if ((debug_mode > 1) && (train_words % 100000 == 0)) {
      printf("%lldK%c", train_words / 1000, 13);
      fflush(stdout);
    }
    i = SearchVocab(word);
    if (i == -1) {
      a = AddWordToVocab(word);
      vocab[a].cn = 1;
    } else vocab[i].cn++;
    if (vocab_size > vocab_hash_size * 0.7) ReduceVocab();
  }
  SortVocab();
  if (debug_mode > 0) {
    printf("Vocab size: %lld\n", vocab_size);
    printf("Words in train file: %lld\n", train_words); 
  }
  file_size = ftell(fin);
  fclose(fin);
}

void SaveVocab() {
  long long i;
  FILE *fo = fopen(save_vocab_file, "wb");
  for (i = 0; i < vocab_size; i++) fprintf(fo, "%s %lld\n", vocab[i].word, vocab[i].cn);
  fclose(fo);
}
vector<vertex_id_t> id2offset;
void ReadVocabFromDegree(vector<vertex_id_t>& degrees){
  vertex_id_t v_num = degrees.size();
  long long a, i = 0;
  char word[MAX_STRING];
  for (a = 0; a < vocab_hash_size; a ++) vocab_hash[a] = -1;
  vocab_size = 0;
  for (vertex_id_t v = 0; v < v_num; v++)
  {
    std::sprintf(word,"%u",v);  // node ID 以字符串的形式存在 vocab 里面。
    a = AddWordToVocab(word);
    vocab[a].cn = degrees[v];
  }
  // 现在vocab 里面存了所有 {nodeId,degree} 的形式。
  SortVocab();
  if (debug_mode > 0) {
    printf("Vocab size: %lld\n", vocab_size);
    printf("Words in train file: %lld\n", train_words);
  }
  id2offset.resize(vocab_size);
  for(vertex_id_t vi = 0; vi<vocab_size; vi++){
    char* endptr;
    vertex_id_t nid = (vertex_id_t)strtoul(vocab[vi].word,&endptr,10);
    id2offset[nid] = vi;
  } 
}

void ReadVocab() {
  long long a, i = 0;
  char c;
  char word[MAX_STRING];
  FILE *fin = fopen(read_vocab_file, "rb");
  if (fin == NULL) {
    printf("Vocabulary file not found\n");
    exit(1);
  }
  for (a = 0; a < vocab_hash_size; a++) vocab_hash[a] = -1;
  vocab_size = 0;
  while (1) {
    ReadWord(word, fin);
    if (feof(fin)) break;
    a = AddWordToVocab(word);
    fscanf(fin, "%lld%c", &vocab[a].cn, &c);
    i++;
  }
  SortVocab();
  if (debug_mode > 0) {
    printf("Vocab size: %lld\n", vocab_size);
    printf("Words in train file: %lld\n", train_words);
  }
  fin = fopen(train_file, "rb");
  if (fin == NULL) {
    printf("ERROR: training data file not found!\n");
    exit(1);
  }
  fseek(fin, 0, SEEK_END);
  file_size = ftell(fin);
  fclose(fin);
}

void InitNet() {
  long long a, b;
  unsigned long long next_random = 1;
  a = posix_memalign((void **)&last_emb, 128, (long long)vocab_size * layer1_size * sizeof(float));
  memset(last_emb,0,(long long)vocab_size * layer1_size * sizeof(float));

  if (last_emb == NULL) {printf("Memory allocation failed\n"); exit(1);}
  
  a = posix_memalign((void **)&syn0, 128, (long long)vocab_size * layer1_size * sizeof(float));
  if (syn0 == NULL) {printf("Memory allocation failed\n"); exit(1);}
  if (hs) {
    a = posix_memalign((void **)&syn1, 128, (long long)vocab_size * layer1_size * sizeof(float));
    if (syn1 == NULL) {printf("Memory allocation failed\n"); exit(1);}
    for (a = 0; a < vocab_size; a++) for (b = 0; b < layer1_size; b++)
      syn1[a * layer1_size + b] = 0;
    checkCUDAerr(hipMalloc((void **)&d_syn1, (long long)vocab_size * layer1_size * sizeof(float)));
    checkCUDAerr(hipMemcpy(d_syn1, syn1, (long long)vocab_size * layer1_size * sizeof(float), hipMemcpyHostToDevice));
  }
  if (negative>0) {
    a = posix_memalign((void **)&syn1neg, 128, (long long)vocab_size * layer1_size * sizeof(float));
    if (syn1neg == NULL) {printf("Memory allocation failed\n"); exit(1);}
    for (a = 0; a < vocab_size; a++) for (b = 0; b < layer1_size; b++)
      syn1neg[a * layer1_size + b] = 0;
    checkCUDAerr(hipMalloc((void **)&d_syn1, (long long)vocab_size * layer1_size * sizeof(float)));
    checkCUDAerr(hipMemcpy(d_syn1, syn1neg, (long long)vocab_size * layer1_size * sizeof(float), hipMemcpyHostToDevice));
  }
  for (a = 0; a < vocab_size; a++) for (b = 0; b < layer1_size; b++) {
    next_random = next_random * (unsigned long long)25214903917 + 11;
    syn0[a * layer1_size + b] = (((next_random & 0xFFFF) / (float)65536) - 0.5) / layer1_size;
  }
  checkCUDAerr(hipMalloc((void **)&d_syn0, (long long)vocab_size * layer1_size * sizeof(float)));
  checkCUDAerr(hipMemcpy(d_syn0, syn0, (long long)vocab_size * layer1_size * sizeof(float), hipMemcpyHostToDevice));

  CreateBinaryTree();
}

void cbowKernel(int *d_sen, int *d_sent_len, float alpha, int cnt_sentence, int reduSize)
{
  int bDim = layer1_size;
  int gDim = cnt_sentence;
  switch(reduSize) {
    case 128: cbow_kernel<64><<<gDim, bDim>>>
              (window, layer1_size, negative, hs, table_size, vocab_size, alpha,
               d_expTable, d_table, d_vocab_codelen, d_vocab_point, d_vocab_code,
               d_sen, d_sent_len, d_syn1, d_syn0);
              break;
    case 256: cbow_kernel<128><<<gDim, bDim>>>
              (window, layer1_size, negative, hs, table_size, vocab_size, alpha,
               d_expTable, d_table, d_vocab_codelen, d_vocab_point, d_vocab_code,
               d_sen, d_sent_len, d_syn1, d_syn0);
              break;
    case 512: cbow_kernel<256><<<gDim, bDim>>>
              (window, layer1_size, negative, hs, table_size, vocab_size, alpha,
               d_expTable, d_table, d_vocab_codelen, d_vocab_point, d_vocab_code,
               d_sen, d_sent_len, d_syn1, d_syn0);
              break;
    default: printf("Can't support on vector size = %lld\n", layer1_size);
             exit(1);
             break;
  }

}

void sgKernel(int *d_sen, int *d_sent_len, int *d_negSample, float alpha, int cnt_sentence, int reduSize)
{
  int bDim= layer1_size;
  int gDim= cnt_sentence;

  if (reuseNeg) { // A sentence share negative samples
    dim3 bDimNeg(32, negative+1, 1);
    switch(layer1_size) {
      case 200: __sgNegReuse<200><<<gDim, bDimNeg>>>
                (window, layer1_size, negative, vocab_size, alpha,
                 d_sen, d_sent_len, d_syn1, d_syn0, d_negSample);
                break;
      case 300: __sgNegReuse<300><<<gDim, bDimNeg>>>
                (window, layer1_size, negative, vocab_size, alpha,
                 d_sen, d_sent_len, d_syn1, d_syn0, d_negSample);
                break;
      default: printf("Can't support on vector size = %lld\n", layer1_size);
               exit(1);
               break;
    }
  } else {
    switch(reduSize) {
      case 128: skip_gram_kernel<64><<<gDim, bDim>>>
                (window, layer1_size, negative, hs, table_size, vocab_size, alpha,
                 d_expTable, d_table, d_vocab_codelen, d_vocab_point, d_vocab_code,
                 d_sen, d_sent_len, d_syn1, d_syn0);
                break;
      case 256: skip_gram_kernel<128><<<gDim, bDim>>>
                (window, layer1_size, negative, hs, table_size, vocab_size, alpha,
                 d_expTable, d_table, d_vocab_codelen, d_vocab_point, d_vocab_code,
                 d_sen, d_sent_len, d_syn1, d_syn0);
                break;
      case 512: skip_gram_kernel<256><<<gDim, bDim>>>
                (window, layer1_size, negative, hs, table_size, vocab_size, alpha,
                 d_expTable, d_table, d_vocab_codelen, d_vocab_point, d_vocab_code,
                 d_sen, d_sent_len, d_syn1, d_syn0);
                break;
      default: printf("Can't support on vector size = %lld\n", layer1_size);
               exit(1);
               break;
    }
  }
}

LR *lr_scheduler;
void TrainModelThread(string data_path)
{
  printf("[ p%d ]=====================Train file %s============\n",my_rank,data_path.c_str());
  long long word, word_count = 0, last_word_count = 0;
  long long local_iter = iter;

  // use in kernel
  int total_sent_len, reduSize= 32;
  int *sen, *sentence_length, *d_sen, *d_sent_len;
  sen = (int *)malloc(MAX_SENTENCE * 100 * sizeof(int));
  sentence_length = (int *)malloc((MAX_SENTENCE + 1) * sizeof(int));

  checkCUDAerr(hipMalloc((void **)&d_sen, MAX_SENTENCE * 100 * sizeof(int)));
  checkCUDAerr(hipMalloc((void **)&d_sent_len, (MAX_SENTENCE + 1) * sizeof(int)));

  int *negSample = (int *)malloc(MAX_SENTENCE * negative * sizeof(int));
  int *d_negSample;
  checkCUDAerr(hipMalloc(&d_negSample, MAX_SENTENCE * negative * sizeof(int)));

  while (reduSize < layer1_size) {
    reduSize *= 2;
  }

  clock_t now;
  strcpy(train_file,data_path.c_str());
  FILE *fi = fopen(train_file, "r");
  if(fi == nullptr) {
    printf("open [%s] fail\n",data_path.c_str());
    throw std::runtime_error("Data file open fail");
  }
  fseek(fi, 0, SEEK_SET);

  while (1) {
    if (word_count - last_word_count > 10000) {
      word_count_actual += word_count - last_word_count;
      last_word_count = word_count;
      if ((debug_mode > 1)) {
        now = clock();
        printf("%cAlpha: %f  Progress: %.2f%%  Words/sec: %.2fk  ", 13, alpha,
            word_count_actual / (float)(iter * train_words + 1) * 100,
            word_count_actual / ((float)(now - start + 1) / (float)CLOCKS_PER_SEC * 1000));
        fflush(stdout);
      }
      // alpha = starting_alpha * (1 - word_count / (float)(iter * train_words + 1));
      // if (alpha < starting_alpha * 0.0001) alpha = starting_alpha * 0.0001;
    }
    total_sent_len = 0;
    sentence_length[0] = 0;
    int cnt_sentence = 0;

    while (cnt_sentence < MAX_SENTENCE) {                           // Read words
      int temp_sent_len = 0;
      char tSentence[MAX_SENTENCE_LENGTH];
      char *wordTok;
      if (feof(fi)) break;
      fgets(tSentence, MAX_SENTENCE_LENGTH + 1, fi);
      wordTok = strtok(tSentence, " \n\r\t");
      while(1) {
        if (wordTok == NULL) {
          word_count++;
          break;
        }
        word = SearchVocab(wordTok);
        wordTok = strtok(NULL, " \n\r\t");
        if (word == -1) continue;
        word_count++;
        if (word == 0) {
          word_count++;
          break;
        }
        if (sample > 0) {
          float ran = (sqrt(vocab[word].cn / (sample * train_words)) + 1) * (sample * train_words) / vocab[word].cn;
          int next_random_t = rand();
          if (ran < (next_random_t & 0xFFFF) / (float)65536) continue;
        }
        sen[total_sent_len] = word;
        total_sent_len++;
        temp_sent_len++;
        if (temp_sent_len >= MAX_SENTENCE_LENGTH) break;
      }
      if (word == 0) {
        word_count++;
        break;
      }
      if (temp_sent_len >= MAX_SENTENCE_LENGTH) break;

      cnt_sentence++;
      sentence_length[cnt_sentence] = total_sent_len;
      if (total_sent_len >= (MAX_SENTENCE - 1) * 20) break;
    }

    if (feof(fi) || (word_count > train_words)) {                   // Initialize for iteration
      word_count_actual += word_count - last_word_count;
      local_iter--;
      if (local_iter == 0) break;
      word_count = 0;
      last_word_count = 0;
      for (int i=0; i<MAX_SENTENCE+1; i++)
        sentence_length[i] = 0;
      total_sent_len = 0;
      fseek(fi, 0, SEEK_SET);
      continue;
    }

    // Negative sampling in advance. A sentence shares negative samples
    for (int i=0; i<cnt_sentence * negative; i++) {
      int randd = rand();
      int tempSample = table[randd % table_size];
      if (tempSample == 0) negSample[i] = randd % (vocab_size - 1) + 1;
      else                 negSample[i] = tempSample;
    }
    checkCUDAerr(hipMemcpy(d_negSample, negSample, cnt_sentence * negative * sizeof(int), hipMemcpyHostToDevice));
    hipError_t cet = hipMemcpy(d_sen, sen, total_sent_len * sizeof(int), hipMemcpyHostToDevice);
    if (hipSuccess != cet)
    {
      printf("%s %d : %s\n", __FILE__, __LINE__, hipGetErrorString(cet));
      printf("copy size: %zu \n",total_sent_len*sizeof(int));
      exit(0);
    }
    checkCUDAerr(hipMemcpy(d_sent_len, sentence_length, (cnt_sentence + 1) * sizeof(int), hipMemcpyHostToDevice));

    if (cbow)
      cbowKernel(d_sen, d_sent_len, alpha, cnt_sentence, reduSize);
    else
      sgKernel(d_sen, d_sent_len, d_negSample, alpha, cnt_sentence, reduSize);
  }
  hipDeviceSynchronize();
  checkCUDAerr(hipMemcpy(syn0, d_syn0, vocab_size * layer1_size * sizeof(float), hipMemcpyDeviceToHost));

  fclose(fi);

  // free memory
  free(sen);
  free(sentence_length);
  free(negSample);
  hipFree(d_sen);
  hipFree(d_sent_len);
  hipFree(d_negSample);
}
vector<vertex_id_t> g_v_degree;

void myIntersectition(const vector<vertex_id_t>& v1,const vector<vertex_id_t>& v2,vector<vertex_id_t>& v_intersection)
{
    int p1=0;
    int p2=0;
    int v1_sz = v1.size();
    int v2_sz = v2.size();
    const vector<vertex_id_t>*long_v;
    const vector<vertex_id_t>*short_v;
    if(v1_sz>v2_sz){
        long_v=&v1;
        short_v=&v2;
    }else{
        long_v=&v2;
        short_v=&v1;
    }
    int max_sz = max(v1_sz,v2_sz);
    int min_sz = min(v1_sz,v2_sz);
    
    int begin = 0;
    if(v1.empty()||v2.empty())return;
    if((*long_v)[max_sz-1]<(*short_v)[0]||((*long_v)[0]>(*short_v)[min_sz-1]))return;
    while(p1<max_sz&&p2<min_sz){
        int offset = 1;
        int last_p = offset;
        if((*short_v)[p2]<((*long_v)[p1])){
            p2++;
            continue;
        }
        while((*long_v)[p1+offset-1]<(*short_v)[p2]){
            offset=offset*2;
            last_p = p1+offset<max_sz?offset:max_sz-p1;
            if(p1+offset>=max_sz)break;
        }
        if((*long_v)[max_sz-1]<(*short_v)[p2]){
            p2++;
            break;
        }
        auto iter = lower_bound(long_v->begin()+(p1+offset/2),long_v->begin()+p1+last_p,(*short_v)[p2]); // 如果在 区间找到了
        int t = iter - long_v->begin();
        if(*iter==(*short_v)[p2]){
            v_intersection.push_back((*short_v)[p2]);
            p2++;
            p1=t++;
        }else{
            p2++;
            p1=(p1+offset/2);
        }
    };   
   
}

float cos_sim(float* v1,float* v2, int dim){
  float dot_product = 0.0;
  float v1_l2 = 0.0, v2_l2 = 0.0;
  for(int d = 0; d < dim; d++){
    dot_product += v1[d] * v2[d];
    v1_l2 += v1[d] * v1[d];
    v2_l2 += v2[d] * v2[d];
  }
  v1_l2 = sqrt(v1_l2);
  v2_l2 = sqrt(v2_l2);
  return dot_product/(v1_l2 * v2_l2);
}
float node_neighbour_average_cos_sim(vertex_id_t v_id,myEdgeContainer*csr){
  float sum_cos_sim = 0.0f;
  int nei_n = csr->adj_lists[v_id].end - csr->adj_lists[v_id].begin;
  // 1. get neighbour set;
  for(auto it = csr->adj_lists[v_id].begin; it < csr->adj_lists[v_id].end; it++){
    vertex_id_t nei = it->neighbour;
    vertex_id_t v_1 = id2offset[v_id];
    vertex_id_t v_2 = id2offset[nei];
    float sim = cos_sim(syn0+v_1 * layer1_size, syn0+v_2*layer1_size,layer1_size);
    sum_cos_sim += sim;
  }
  return sum_cos_sim / nei_n;
}

float find_supernode_topK_accurancy(float p,int k,myEdgeContainer*csr){
  float top_sum = 0;
  for(vertex_id_t v_i = 0; v_i < vocab_size*0.03; v_i ++){
    vector<std::pair<float,vertex_id_t>> supernode_sim;
    for(vertex_id_t v_j = 0; v_j < vocab_size * p; v_j ++){
      float sim = cos_sim(syn0+v_i * layer1_size, syn0+v_j*layer1_size,layer1_size);
      supernode_sim.push_back({sim,v_j});
    }
    sort(supernode_sim.begin(),supernode_sim.end(),[](std::pair<float,vertex_id_t>&p1,std::pair<float,vertex_id_t>&p2){
        return p1.first > p2.first;
        });
    vector<vertex_id_t> selected_topK(k);
    for(int i = 0 ;i< k; i++){
      // selected_topK[i] = supernode_sim[i].second;
      selected_topK[i] = supernode_sim[i].second ; 
    }
    vector<vertex_id_t> real_neighbor;
    for(auto it = csr->adj_lists[v_i].begin; it < csr->adj_lists[v_i].end; it++){
      real_neighbor.push_back(it->neighbour);
    }
    sort(real_neighbor.begin(),real_neighbor.end());
    sort(selected_topK.begin(),selected_topK.end());
    vector<vertex_id_t>result_set;
    myIntersectition(selected_topK, real_neighbor, result_set);
    top_sum += result_set.size();
  }
  return top_sum/ (float)(k * vocab_size *p);
}
void TrainModel(SyncQueue& taskq,myEdgeContainer*csr) {
  printf("==========================Train Model In=====================\n");
  long a, b, c, d;
  FILE *fo;
  starting_alpha = alpha;
  ReadVocabFromDegree(g_v_degree);
  printf("========================Read Vocab ok=======================\n");
  printf("vocab_size: %lu\n",vocab_size);

  // for(size_t i = 0; i < vocab_size * 0.10;i++){
  //   printf("id: %s, degree: %ld\n",vocab[i].word,vocab[i].cn);
  // }

  vector<float> H;
  float delta_H;
  

  // if (read_vocab_file[0] != 0) ReadVocab(); else LearnVocabFromTrainFile();
  // if (save_vocab_file[0] != 0) SaveVocab();
  if (output_file[0] == 0) printf("[ Warning ] output file missing\n");
  if (output_file[0] == 0) return;
  InitNet();
  printf("[ %d ] InitNet Success\n",my_rank);
  if (hs > 0) InitVocabStructCUDA();
  if (negative > 0) InitUnigramTable();

  start = clock();
  srand(time(NULL));

  printf("==========init success================\n");
  float* kl = new float;
  // lr_scheduler = new  FixedLR(0.025);
  // lr_scheduler = new  StepDecayLR(0.025,0.5,3);
  lr_scheduler = new ExponentialDecayLR(0.025,0.1);

  // TrainModelThread("./out/tmp-0-1.txt");
  // float acc = find_supernode_topK_accurancy(0.01,10,csr);
  //
  // cout << "find super node topK acc: " << acc << endl;

  FILE* f_nei_cos_sim = fopen("neighbour_average_cos_sim.txt","w");
  vector<vector<float>>node_neighbour_average_cos_sim_array(vocab_size);
  int n2 = 2;
  while(n2++ < 30){
    char fc[100];
    sprintf(fc,"./out/tmp-0-%d.txt",n2);
    alpha = lr_scheduler->get_lr();
    TrainModelThread(fc);
    std::cout << std::endl;
    for(vertex_id_t v = 0;v < vocab_size; v++){
      float s = node_neighbour_average_cos_sim(v,csr);
      node_neighbour_average_cos_sim_array[v].push_back(s);
    }
  }
  for(vertex_id_t i = 0; i < vocab_size; i++){
    for(int j = 0; j < node_neighbour_average_cos_sim_array[i].size();j++){
      fprintf(f_nei_cos_sim, "%.3f ",node_neighbour_average_cos_sim_array[i][j]);
    }
    fprintf(f_nei_cos_sim,"\n");
  }
  fclose(f_nei_cos_sim);
  float p = 0.6;
  vector<bool> flag(vocab_size,true);
  int sum_count = 0;
  for(int s = 0;s < node_neighbour_average_cos_sim_array[0].size();s++){
    int count = 0;
    for(vertex_id_t v = 0;v < vocab_size;v++){
      if(node_neighbour_average_cos_sim_array[s][v] > p && flag[v] == true){
        count++;
        flag[v] = false;
      }
    }
    cout <<count <<" ";
    sum_count+=count;
  }
  cout << " sum: "<< sum_count << endl;
  return;

  FILE* f_topk = fopen("super_topK.txt","w");
  int n1 = 2;
  while(n1++<30){
    char fc[100];
    sprintf(fc,"./out/tmp-0-%d.txt",n1);
    alpha = lr_scheduler->get_lr();
    TrainModelThread(fc);
    std::cout << std::endl;
    float acc = find_supernode_topK_accurancy(0.01,10,csr);
    cout << "find super node topK acc: " << acc << endl;
    fprintf(f_topk,"%f\n",acc);
  }
  fclose(f_topk);
  return;

  // [Test]
  
  //compute_kl_from_emb(last_emb,syn0, kl,tmpN,layer1_size);
  //std::cout << std::endl;
  //checkCUDAerr(hipDeviceSynchronize()); 
  //printf("[ %d ] COMPUTE KL From Emb : %.f\n",my_rank,*kl);

  // [End Test]

 //  FILE* f_rel_ent = fopen("rel_ent.txt","w");
 //  FILE* f_delta_ent = fopen("delta_ent.txt","w");
 // int n = 2;
 //  while(n++<30){
 //    char fc[100];
 //    sprintf(fc,"./out/tmp-0-%d.txt",n);
 //    TrainModelThread(fc);
 //    cout << endl;
 //    // termination judgment
 //    compute_kl_from_emb(last_emb,syn0, kl,vocab_size * 0.1,layer1_size);
 //    checkCUDAerr(hipDeviceSynchronize()); 
 //    printf("[ %d ] COMPUTE KL from emb : %.f\n",my_rank,*kl);
 //    fprintf(f_rel_ent,"%s: %f\n",fc,*kl);
 //    memcpy(last_emb,syn0,(long long)vocab_size * layer1_size * sizeof(float));
 //  }
 //  fclose(f_delta_ent);
 //  fclose(f_rel_ent);
 //  printf("=============Task over | Calculate delta H ===========\n");
  
  
  hipFree(d_table);
  hipFree(d_syn1);
  hipFree(d_syn0);
  hipFree(d_vocab_codelen);
  hipFree(d_vocab_point);
  hipFree(d_vocab_code);

  if(my_rank != 0) return;

  std::chrono::steady_clock::time_point t1 = std::chrono::steady_clock::now();

  fo = fopen(output_file, "wb");
  if(fo == NULL) printf("[ %d ] [%s] open fail\n",output_file);
  if (classes == 0) {	
    // Save the word vectors
    fprintf(fo, "%lld %lld\n", vocab_size, layer1_size);
    for (a = 0; a < vocab_size; a++) {
      fprintf(fo, "%s ", vocab[a].word);
      if (binary) for (b = 0; b < layer1_size; b++) fwrite(&syn0[a * layer1_size + b], sizeof(float), 1, fo);
      else for (b = 0; b < layer1_size; b++) fprintf(fo, "%lf ", syn0[a * layer1_size + b]);
      fprintf(fo, "\n");
    }
  } else {
    // Run K-means on the word vectors
    int clcn = classes, iter = 10, closeid;
    int *centcn = (int *)malloc(classes * sizeof(int));
    int *cl = (int *)calloc(vocab_size, sizeof(int));
    float closev, x;
    float *cent = (float *)calloc(classes * layer1_size, sizeof(float));

    for (a = 0; a < vocab_size; a++) cl[a] = a % clcn;
    for (a = 0; a < iter; a++) {
      for (b = 0; b < clcn * layer1_size; b++) cent[b] = 0;
      for (b = 0; b < clcn; b++) centcn[b] = 1;
      for (c = 0; c < vocab_size; c++) {
        for (d = 0; d < layer1_size; d++) cent[layer1_size * cl[c] + d] += syn0[c * layer1_size + d];
        centcn[cl[c]]++;
      }
      for (b = 0; b < clcn; b++) {
        closev = 0;
        for (c = 0; c < layer1_size; c++) {
          cent[layer1_size * b + c] /= centcn[b];
          closev += cent[layer1_size * b + c] * cent[layer1_size * b + c];
        }
        closev = sqrt(closev);
        for (c = 0; c < layer1_size; c++) cent[layer1_size * b + c] /= closev;
      }
      for (c = 0; c < vocab_size; c++) {
        closev = -10;
        closeid = 0;
        for (d = 0; d < clcn; d++) {
          x = 0;
          for (b = 0; b < layer1_size; b++) x += cent[layer1_size * d + b] * syn0[c * layer1_size + b];
          if (x > closev) {
            closev = x;
            closeid = d;
          }
        }
        cl[c] = closeid;
      }
    }

    // Save the K-means classes
    for (a = 0; a < vocab_size; a++) fprintf(fo, "%s %d\n", vocab[a].word, cl[a]);


    free(centcn);
    free(cent);
    free(cl);
  }
  std::chrono::steady_clock::time_point t2 = std::chrono::steady_clock::now();
  std::chrono::duration<double> time_span = std::chrono::duration_cast<std::chrono::duration<double>>(t2-t1);
  std::cout<<"[ "<<my_rank<<" ] Save Embedding: " <<time_span.count() << " s" <<std::endl;
  fclose(fo);
}

int ArgPos(char *str, int argc, char **argv) {
  int a;
  for (a = 1; a < argc; a++) if (!strcmp(str, argv[a])) {
    if (a == argc - 1) {
      printf("Argument missing for %s\n", str);
      exit(1);
    }
    return a;
  }
  return -1;
}
int train_corpus_cuda(int argc, char **argv,const vector<vertex_id_t>& degrees,SyncQueue& corpus_q,int _my_rank,myEdgeContainer* csr) {

  printf("No.10%: %llu\n",degrees[degrees.size() * 0.03]);
  // test area 

  // test __global__ void cosine_similarity_kernel(float *d_vectors, float *d_result, int v, int dim){
  float h_vec[] = {1,2,3,1,4,1};
  float h_vec2[] = {1,1,1,1,2,1};
  float *d_vec;
  float *d_result;
  int v = 2;
  int dim = 3;
  float* h_result = new float[v * dim];

  checkCUDAerr(hipMalloc((void **)&d_result, (long long)v * dim * sizeof(float)));
  checkCUDAerr(hipMalloc((void **)&d_vec, (long long)v * dim * sizeof(float)));
  checkCUDAerr(hipMemcpy(d_vec, h_vec, (long long)v * dim * sizeof(float), hipMemcpyHostToDevice));

	dim3 blockSize(16,16);
	dim3 gridSize((v+blockSize.x-1) / blockSize.x,(v+blockSize.y-1)/blockSize.y);
  cosine_similarity_kernel<<<gridSize,blockSize>>>(d_vec,d_result,v,dim);

  checkCUDAerr(hipMemcpy(h_result, d_result, (long long)v * dim * sizeof(float), hipMemcpyDeviceToHost));

  for(int i = 0; i < v * v; i++){
    std::cout << h_result[i] << " ";
  }
  std::cout << std::endl;

  checkCUDAerr(hipMemcpy(d_vec, h_vec2, (long long)v * dim * sizeof(float), hipMemcpyHostToDevice));
  cosine_similarity_kernel<<<gridSize,blockSize>>>(d_vec,d_result,v,dim);
  checkCUDAerr(hipMemcpy(h_result, d_result, (long long)v * dim * sizeof(float), hipMemcpyDeviceToHost));
  // TEST __global__ void compute_kl_divergence_kernel(float *d_A,float *d_B, float *d_result, int v){
  for(int i = 0; i < v * v; i++){
    std::cout << h_result[i] << " ";
  }
  std::cout << std::endl;
  
  float h_A[] = {1,0.942809,0.942809,1 };
  float h_B[] = {1,0.755929,0.755929,1 };
  float *d_A,*d_B,*d_res;
  float *h_res = new float;
  
  checkCUDAerr(hipMalloc((void **)&d_A, (long long)v * v * sizeof(float)));
  checkCUDAerr(hipMalloc((void **)&d_B, (long long)v * v * sizeof(float)));
  checkCUDAerr(hipMemcpy(d_A, h_A, (long long)v * v * sizeof(float), hipMemcpyHostToDevice));
  checkCUDAerr(hipMemcpy(d_B, h_B, (long long)v * v * sizeof(float), hipMemcpyHostToDevice));
  checkCUDAerr(hipMalloc((void **)&d_res,sizeof(float)));

  int blockSize2 = 256;
  int gridSize2 = (v * v + blockSize2 - 1) / blockSize2;
  compute_kl_divergence_kernel<<<gridSize2,blockSize2>>>(d_A,d_B,d_res,v);
  checkCUDAerr(hipMemcpy(h_res,d_res,sizeof(float),hipMemcpyDeviceToHost));

  std::cout<< "KL: " << *h_res<< std::endl;
  

  // TEST void  compute_kl_from_emb(float *emb1, float *emb2,float* h_result, int v,int dim){
  float h_test1[] = {1,1,1,1,2,1};
  float h_test2[] = {1,2,3,1,4,1};

  float* h_kl = new float;
  compute_kl_from_emb(h_test1,h_test2,h_kl,2,3);
  printf("[TEST] GET Kl From Emb: %f\n",*h_kl);

  // end test area

  my_rank = _my_rank;
  g_v_degree.assign(degrees.begin(), degrees.end());
  printf("train_corpus_Cuda calling!!!!\n");
  int i;
  if (argc == 1) {
    printf("WORD VECTOR estimation toolkit v 0.1c\n\n");
    printf("Options:\n");
    printf("Parameters for training:\n");
    printf("\t-train <file>\n");
    printf("\t\tUse text data from <file> to train the model\n");
    printf("\t-output <file>\n");
    printf("\t\tUse <file> to save the resulting word vectors / word clusters\n");
    printf("\t-size <int>\n");
    printf("\t\tSet size of word vectors; default is 100\n");
    printf("\t-window <int>\n");
    printf("\t\tSet max skip length between words; default is 5\n");
    printf("\t-sample <float>\n");
    printf("\t\tSet threshold for occurrence of words. Those that appear with higher frequency in the training data\n");
    printf("\t\twill be randomly down-sampled; default is 1e-3, useful range is (0, 1e-5)\n");
    printf("\t-hs <int>\n");
    printf("\t\tUse Hierarchical Softmax; default is 0 (not used)\n");
    printf("\t-negative <int>\n");
    printf("\t\tNumber of negative examples; default is 5, common values are 3 - 10 (0 = not used)\n");
    printf("\t-reuse-neg <int>\n");
    printf("\t\tA sentence share a negative sample set; (0 = not used / 1 = used)\n");

    printf("\t-iter <int>\n");
    printf("\t\tRun more training iterations (default 5)\n");
    printf("\t-min-count <int>\n");
    printf("\t\tThis will discard words that appear less than <int> times; default is 5\n");
    printf("\t-alpha <float>\n");
    printf("\t\tSet the starting learning rate; default is 0.025 for skip-gram and 0.05 for CBOW\n");
    printf("\t-classes <int>\n");
    printf("\t\tOutput word classes rather than word vectors; default number of classes is 0 (vectors are written)\n");
    printf("\t-debug <int>\n");
    printf("\t\tSet the debug mode (default = 2 = more info during training)\n");
    printf("\t-binary <int>\n");
    printf("\t\tSave the resulting vectors in binary moded; default is 0 (off)\n");
    printf("\t-save-vocab <file>\n");
    printf("\t\tThe vocabulary will be saved to <file>\n");
    printf("\t-read-vocab <file>\n");
    printf("\t\tThe vocabulary will be read from <file>, not constructed from the training data\n");
    printf("\t-cbow <int>\n");
    printf("\t\tUse the continuous bag of words model; default is 1 (use 0 for skip-gram model)\n");
    printf("\nExamples:\n");
    printf("./word2vec -train data.txt -output vec.txt -size 200 -window 5 -sample 1e-4 -negative 5 -hs 0 -binary 0 -cbow 1 -iter 3\n\n");
    return 0;
  }

  output_file[0] = 0;
  save_vocab_file[0] = 0;
  read_vocab_file[0] = 0;
  if ((i = ArgPos((char *)"-size", argc, argv)) > 0) layer1_size = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-train", argc, argv)) > 0) strcpy(train_file, argv[i + 1]);
  if ((i = ArgPos((char *)"-save-vocab", argc, argv)) > 0) strcpy(save_vocab_file, argv[i + 1]);
  if ((i = ArgPos((char *)"-read-vocab", argc, argv)) > 0) strcpy(read_vocab_file, argv[i + 1]);
  if ((i = ArgPos((char *)"-debug", argc, argv)) > 0) debug_mode = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-binary", argc, argv)) > 0) binary = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-cbow", argc, argv)) > 0) cbow = atoi(argv[i + 1]);
  if (cbow) alpha = 0.05;
  if ((i = ArgPos((char *)"-alpha", argc, argv)) > 0) alpha = atof(argv[i + 1]);
  if ((i = ArgPos((char *)"-emb_output", argc, argv)) > 0) strcpy(output_file, argv[i + 1]);
  if ((i = ArgPos((char *)"-window", argc, argv)) > 0) window = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-sample", argc, argv)) > 0) sample = atof(argv[i + 1]);
  if ((i = ArgPos((char *)"-hs", argc, argv)) > 0) hs = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-negative", argc, argv)) > 0) negative = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-iter", argc, argv)) > 0) iter = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-min-count", argc, argv)) > 0) min_count = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-classes", argc, argv)) > 0) classes = atoi(argv[i + 1]);
  if ((i = ArgPos((char *)"-reuse-neg", argc, argv)) > 0) reuseNeg = atoi(argv[i + 1]);

  vocab = (struct vocab_word *)calloc(vocab_max_size, sizeof(struct vocab_word));
  vocab_hash = (int *)calloc(vocab_hash_size, sizeof(int));
  expTable = (float *)malloc((EXP_TABLE_SIZE + 1) * sizeof(float));

  for (i = 0; i < EXP_TABLE_SIZE; i++) {
    expTable[i] = exp((i / (float)EXP_TABLE_SIZE * 2 - 1) * MAX_EXP); // Precompute the exp() table
    expTable[i] = expTable[i] / (expTable[i] + 1);                   // Precompute f(x) = x / (x + 1)
  }
  checkCUDAerr(hipMalloc((void **)&d_expTable, (EXP_TABLE_SIZE + 1) * sizeof(float)));
  checkCUDAerr(hipMemcpy(d_expTable, expTable, (EXP_TABLE_SIZE + 1) * sizeof(float), hipMemcpyHostToDevice));

  TrainModel(corpus_q,csr);

  // memory free
  free(vocab_codelen);
  free(vocab_point);
  free(vocab_code);
  free(table);
  free(syn0);
  free(syn1);
  free(syn1neg);
  free(vocab);
  free(vocab_hash);
  free(expTable);
  hipFree(d_expTable);
  free(last_emb);

  return 0;
}
